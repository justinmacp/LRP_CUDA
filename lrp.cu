#include "hip/hip_runtime.h"
#include <stdlib.h>

#include "hipDNN.h"

__global__ void forward_pass(float *pInputs, float *pOutputs) {
	int Inx = blockIdx.x, Iny = blockIdx.y, 
	}

int lrp() {
	kernel = get_parameter(weight_NCHW_Name128, 9*128*128);
	bnBias = get_parameter(bnBiasName128, 128);
	bnScale = get_parameter(bnScaleName128, 128);
	float* eMean = get_parameter(eMeanName128, 128);
	float* eVar = get_parameter(eVarName128, 128);
	float *l_eMean, *l_eVar;
	nInput = 16*16*128, nOutput = 14*14*128, nWeights = 3*3*128*128, nBias = 128;

	hipMalloc((void **) &output, nOutput<<2);
	hipMalloc((void **) &l_weights, nWeights<<2);
	hipMalloc((void **) &l_bias, nBias<<2);
	hipMemcpy(l_weights, kernel, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bias, bias, nBias<<2, hipMemcpyHostToDevice);

	hipMalloc((void **) &l_eMean, nBias<<2);
	hipMalloc((void **) &l_eVar, nBias<<2);
	hipMemcpy(l_bnBias, bnBias, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bnScale, bnScale, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_eMean, eMean, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_eVar, eVar, nBias<<2, hipMemcpyHostToDevice);

	hipMemset((void *) output, 0, nOutput<<2);

	float tmp_cudnn[nOutput];


	/*  2. cuDNN preparation  */
	hipdnnStatus_t status;
	float one = 1.0, zero = 0.0;
	int size;

	hipdnnHandle_t handle;
	status = hipdnnCreate(&handle);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed1\n");

	hipdnnTensorDescriptor_t xdesc, ydesc, bdesc;
	hipdnnFilterDescriptor_t wdesc; // HIPDNN_TENSOR_NHWC, HIPDNN_TENSOR_NCHW
	status = hipdnnCreateTensorDescriptor(&xdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed2\n");
	status = hipdnnSetTensor4dDescriptor(xdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 16, 16);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed3\n");
	status = hipdnnCreateTensorDescriptor(&ydesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed4\n");
	status = hipdnnSetTensor4dDescriptor(ydesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 14, 14);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5\n");
	status = hipdnnCreateFilterDescriptor(&wdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed6\n");
	status = hipdnnSetFilter4dDescriptor(wdesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 128, 128, 3, 3);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed7\n");
	status = hipdnnCreateTensorDescriptor(&bdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed8\n");
	status = hipdnnSetTensor4dDescriptor(bdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed9\n");
	hipdnnConvolutionDescriptor_t conv_desc;
	status = hipdnnCreateConvolutionDescriptor(&conv_desc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed10\n");
	status = hipdnnSetConvolution2dDescriptor(conv_desc, 0,0, 1,1,1,1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT); //HIPDNN_CONVOLUTION
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed11\n");

	hipdnnActivationDescriptor_t act_desc;
	status = hipdnnCreateActivationDescriptor(&act_desc);  
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed12\n");
	status = hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed13\n");

	hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc;
	status = hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVarDesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed14\n");
	status = hipdnnSetTensor4dDescriptor(bnScaleBiasMeanVarDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 128, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed15\n");

	hipdnnConvolutionFwdAlgo_t algo = (hipdnnConvolutionFwdAlgo_t)6;

	status = hipdnnGetConvolutionForwardWorkspaceSize(handle,
	   xdesc,
	   wdesc,
	   conv_desc,
	   ydesc,
	   algo,
	   (size_t *)&(size));

	float *extra;
	hipMalloc((void **) &extra, size);


	/*  3. Computing  */
	nT1_cudnn = getTimeMicroseconds64();

	status = hipdnnConvolutionForward(handle, &one,
		xdesc, input, wdesc, l_weights, 
		conv_desc, algo, 
		extra, size, &zero,
		ydesc, output);
	if (status != HIPDNN_STATUS_SUCCESS) printf("No Success1\n");

	status = hipdnnActivationForward(handle, act_desc, &one,
		ydesc, output, &zero,
		ydesc, output);
	if (status != HIPDNN_STATUS_SUCCESS) printf("No Success3\n");

	hipDeviceSynchronize();
	nT2_cudnn = getTimeMicroseconds64();
	printf("cuDNN TotalTime = %d us\n", nT2_cudnn-nT1_cudnn);


	/*  4. Copy back and free  */
	s = hipMemcpy(tmp_cudnn, output, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));

	hipFree(extra);
	hipFree(input);
	hipFree(output);
	hipFree(l_weights);
	hipFree(l_bias);

	free(bias);
	free(kernel);
	free(input_);

	return (nT2_cudnn-nT1_cudnn);
}
