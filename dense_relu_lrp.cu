
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>


#define IN_FEATS 200
#define OUT_CLASSES 10
#define W_SIZE IN_FEATS * OUT_CLASSES


uint64_t getTimeMicroseconds64()
{
  uint64_t nTime;
  struct timespec tSpec;
  clock_gettime(CLOCK_REALTIME, &tSpec);
  nTime = (uint64_t)tSpec.tv_nsec / 1000;
  return nTime;
}


__global__ void fwd_perc(float *in, float *out, float *weights, float *activations, float *activation_sum)
{
  int b = blockIdx.x;
  int t = threadIdx.x;
  __shared__ float z[IN_FEATS], sum_z;
  sum_z = 0;
  __syncthreads();
  z[t] = in[t] * weights[b * IN_FEATS + t];
  atomicAdd(&sum_z, z[t]);
  __syncthreads();
  activation_sum[b] = sum_z;
  activations[b * IN_FEATS + t ] = z[t];
  if (sum_z < 0) { out[b] = 0; } else { out[b] = sum_z; }
}


__global__ void lrp_perc(float *out, float *relevance, float *activations, float *activation_sum)
{
  int b = blockIdx.x;
  int t = threadIdx.x;
  __shared__ float z[OUT_CLASSES], rel, sum_z[OUT_CLASSES], r_m[OUT_CLASSES];
  z[t] = activations[t * IN_FEATS + b];
  rel = 0;
  sum_z[t] = activation_sum[t];
  r_m[t] = out[t];
  __syncthreads();
  atomicAdd(&rel, z[t] * r_m[t] / sum_z[t]);
  __syncthreads();
  relevance[b] = rel;
}


void lrp_perc_gm(float *in, float *out, float *relevance, float *weights, float *activations, float *activation_sum, int n, int m)
{
  for (int j = 0; j < m; j++) {
    for (int i_prime = 0; i_prime < n; i_prime++) {
      activations[j * n + i_prime] = in[i_prime] * weights[j * n + i_prime];
      activation_sum[j] += activations[j * n + i_prime];
    }
    if (activation_sum[j] < 0) { out[j] = 0; } else { out[j] = activation_sum[j]; }
  }
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < m; j++) {
      relevance[i] += (activations[j * n + i] * out[j]) / activation_sum[j];
    }
  }
}


int main(void)
{
  uint64_t dT1 = 0, dT2 = 0, hT1 = 0, hT2 = 0;
  float input[IN_FEATS], golden_out[OUT_CLASSES], cuda_out[OUT_CLASSES], weights[W_SIZE], golden_relevance[IN_FEATS], cuda_relevance[IN_FEATS], golden_activations[W_SIZE], cuda_activations[W_SIZE], golden_asum[OUT_CLASSES], cuda_asum[OUT_CLASSES];
  hipError_t s;
  
  // initialize variables on host
  for (int i = 0; i < IN_FEATS; i++) {
    input[i] = rand() % 10;
    golden_relevance[i] = 0;
    cuda_relevance[i] = 0;
    for (int j = 0; j < OUT_CLASSES; j++) {
      weights[j * IN_FEATS + i] = rand() % 10;
      golden_activations[j * IN_FEATS + i] = 0;
      cuda_activations[j * IN_FEATS + i] = 0;
    }
  }
  for (int i = 0; i < OUT_CLASSES; i++) {
    golden_out[i] = 0;
    cuda_out[i] = 0;
    golden_asum[i] = 0;
    cuda_asum[i] = 0;
  }

  // allocating memory for variables for device
  float *input_, *weights_, *output_, *relevance_, *activations_, *asum_;
  hipMalloc(&input_, IN_FEATS * sizeof(float)); 
  hipMalloc(&weights_, W_SIZE * sizeof(float)); 
  hipMalloc(&output_, OUT_CLASSES * sizeof(float));
  hipMalloc(&relevance_, IN_FEATS * sizeof(float));
  hipMalloc(&activations_, W_SIZE * sizeof(float)); 
  hipMalloc(&asum_, OUT_CLASSES * sizeof(float));
  
  // run version with static shared memory
  hipMemcpy(input_, input, IN_FEATS * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(weights_, weights, W_SIZE *sizeof(float), hipMemcpyHostToDevice);
  hipMemset(output_, 0, OUT_CLASSES * sizeof(float));
  hipMemset(relevance_, 0, IN_FEATS * sizeof(float));
  hipMemset(activations_, 0, W_SIZE * sizeof(float));
  hipMemset(asum_, 0, OUT_CLASSES * sizeof(float));

  // run cuda kernel and host function and compare the results
  hT1 = getTimeMicroseconds64();
  lrp_perc_gm(input, golden_out, golden_relevance, weights, golden_activations, golden_asum, IN_FEATS, OUT_CLASSES);
  hT2 = getTimeMicroseconds64();
  dT1 = getTimeMicroseconds64();
  fwd_perc<<<OUT_CLASSES, IN_FEATS>>>(input_, output_, weights_, activations_, asum_);
  lrp_perc<<<IN_FEATS, OUT_CLASSES>>>(output_, relevance_, activations_, asum_);
  s = hipDeviceSynchronize();
  dT2 = getTimeMicroseconds64();
  printf("%s\n", hipGetErrorName(s));

  // relvance
  printf("### RELEVANCE ###\n");
  s = hipMemcpyAsync(cuda_relevance, relevance_, IN_FEATS * sizeof(float), hipMemcpyDeviceToHost);
  printf("%s\n", hipGetErrorName(s));
  for (int i = 0; i < IN_FEATS; i++) {
    if (golden_relevance[i] != cuda_relevance[i]) {
      printf("Error: golden_relevance[%d]!=cuda_relevance[%d] (%f, %f)\n", i, i, golden_relevance[i], cuda_relevance[i]);
    }
  }

  // out
  printf("### OUT ###\n");
  s = hipMemcpy(cuda_out, output_, OUT_CLASSES * sizeof(float), hipMemcpyDeviceToHost);
  printf("%s\n", hipGetErrorName(s));
  for (int i = 0; i < OUT_CLASSES; i++) {
    if (golden_out[i] != cuda_out[i]) {
      printf("Error: golden_out[%d]!=cuda_out[%d] (%f, %f)\n", i, i, golden_out[i], cuda_out[i]);
    }
  }

  // activations
  printf("### ACTIVATIONS ###\n");
  s = hipMemcpy(cuda_activations, activations_, W_SIZE * sizeof(float), hipMemcpyDeviceToHost);
  printf("%s\n", hipGetErrorName(s));
  for (int i = 0; i < W_SIZE; i++) {
    if (golden_activations[i] != cuda_activations[i]) {
      printf("Error: golden_activations[%d]!=cuda_activations[%d] (%f, %f)\n", i, i, golden_activations[i], cuda_activations[i]);
    }
  }

  // asum
  printf("### ASUM ###\n");
  s = hipMemcpy(cuda_asum, asum_, OUT_CLASSES * sizeof(float), hipMemcpyDeviceToHost);
  printf("%s\n", hipGetErrorName(s));
  for (int i = 0; i < OUT_CLASSES; i++) {
    if (golden_asum[i] != cuda_asum[i]) {
      printf("Error: golden_asum[%d]!=cuda_asum[%d] (%f, %f)\n", i, i, golden_asum[i], cuda_asum[i]);
    }
  }
 
  printf("GPU time: %lu, \tCPU time: %lu\n", (dT2 - dT1) << 16, (hT2 - hT1) << 16);

  hipFree(input_); 
  hipFree(weights_); 
  hipFree(output_);
  hipFree(relevance_);
  hipFree(activations_); 
  hipFree(asum_);
}
